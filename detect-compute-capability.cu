
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_runtime_api.h"

using namespace std;

int main(int argc, char ** argv) { 
    int count;
    hipGetDeviceCount(&count); 
    if(count == 0) { 
        std::cerr << "Could not find a CUDA device";
        return 1;
    }
    if(count != 1) { 
        std::cerr << "Warning: Expected exactly one CUDA device, got " << count; 
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); 

    std::cout << prop.major << prop.minor;
    return 0;
}